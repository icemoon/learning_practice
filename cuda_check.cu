
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

void _cudaCheck(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(err)
            << file << line << endl;
        exit(-1);

    }

}
#define cudaCheck(ans) { _cudaCheck((ans), __FILE__, __LINE__);  }


__global__  void add(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

}


int main()
{
    const int N = 16;

    int a[N] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
    int b[N] = {16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31};

    const size_t sz = N * sizeof(int);
    int *da;
    cudaCheck(hipMalloc(&da, sz));
    int *db;
    cudaCheck(hipMalloc(&db, sz));
    int *dc;
    cudaCheck(hipMalloc(&dc, sz));

    cudaCheck(hipMemcpy(da, a, sz, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(db, b, sz, hipMemcpyHostToDevice));

    add<<<1, N>>>(da, db, dc);
    cudaCheck(hipGetLastError());

    int c[N];
    cudaCheck(hipMemcpy(c, dc, sz, hipMemcpyDeviceToHost));

    cudaCheck(hipFree(da));
    cudaCheck(hipFree(db));
    cudaCheck(hipFree(dc));

    for (unsigned int i = 0 ; i < N; ++i) {
        cout << c[i] << " ";

    }
    cout << endl;
    return 0;

}
